#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    const int N = 2; // 2x2 matrices
    const int SIZE = N * N * sizeof(float);

    // Host matrices
    float h_A[N*N] = {1.0f, 2.0f, 3.0f, 4.0f};
    float h_B[N*N] = {5.0f, 6.0f, 7.0f, 8.0f};
    float h_C[N*N] = {0.0f};

    // Device matrices
    float *d_A, *d_B, *d_C;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, SIZE);
    hipMalloc((void**)&d_B, SIZE);
    hipMalloc((void**)&d_C, SIZE);

    // Copy host matrices to device
    hipMemcpy(d_A, h_A, SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, SIZE, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);

    // Launch kernel

    // Synchronize to make sure the kernel has finished
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_C, d_C, SIZE, hipMemcpyDeviceToHost);

    // Print the result
    printf("Result matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_C[i*N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
